//
// Created by Jackson Michael on 3/5/20.
//

#include "devicelist.cuh"

#include <stdexcept>

#include <hip/hip_runtime.h>

namespace cditma {
namespace cuda {

DeviceList::DeviceList()
{
    int num_gpus;
    hipError_t status = hipGetDeviceCount(&num_gpus);
    if(status != hipSuccess) {
        throw std::runtime_error("Failed to ger the number of CUDA devices, with code "
                                 + std::to_string(status));
    }
    cuda_devices_.resize(num_gpus);
}

const int* DeviceList::GetDevices() const
{
    // Return the list of available cuda devices
    return cuda_devices_.data();
}

size_t DeviceList::Size() const
{
    return cuda_devices_.size();
}
int* DeviceList::GetMutableDevices()
{
    return &(*cuda_devices_.begin());
}

} // namespace cuda
} // namespace cditma