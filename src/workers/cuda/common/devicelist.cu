//
// Created by Jackson Michael on 3/5/20.
//

#include "devicelist.cuh"

#include <stdexcept>

#include <hip/hip_runtime.h>

namespace cditma {
namespace cuda {

DeviceList::DeviceList()
{
    // Get the number of GPU devices
    int num_gpus;
    hipError_t status = hipGetDeviceCount(&num_gpus);
    if(status != hipSuccess) {
        throw std::runtime_error("Failed to ger the number of CUDA devices, with code "
                                 + std::to_string(status));
    }

    // Create the vector of devices, with each element equal to the device id (ie the index)
    cuda_devices_.resize(num_gpus);
    for (size_t id=0; id < cuda_devices_.size(); id++) {
        cuda_devices_[id] = id;
    }
}

const int* DeviceList::GetDevices() const
{
    // Return the list of available cuda devices
    return cuda_devices_.data();
}

size_t DeviceList::Size() const
{
    return cuda_devices_.size();
}
int* DeviceList::GetMutableDevices()
{
    return &(*cuda_devices_.begin());
}

} // namespace cuda
} // namespace cditma