//
// Created by Jackson Michael on 1/5/20.
//

#include "cuffthandle.cuh"

#include <iostream>

#include <hipfft/hipfftXt.h>
#include <thrust/device_vector.h>

#include "cufft_error.h"

namespace cditma {
namespace cuda {

using cditma::cuda::cufft::throw_on_error;

CUFFTHandle::CUFFTHandle(DeviceList& devices) : handle_(std::make_unique<hipfftHandle>())
{
    // Perform the basic initialisation of the fft execution plan
    throw_on_error(hipfftCreate(handle_.get()));

    // Point the newly created plan at the list of provided devices
    throw_on_error(hipfftXtSetGPUs(*handle_, devices.Size(), devices.GetMutableDevices()));

    // Create the worksizes for each GPU
    worksizes = new size_t[devices.Size()];
}

CUFFTHandle::~CUFFTHandle()
{
    delete[] worksizes;
    hipfftDestroy(*handle_);
}

CUFFTVec<float> CUFFTHandle::FFT(const CUFFTVec<float>& in_data)
{
    size_t num_samples = in_data.size() / 2;

    // Make the plan now that we know the size of the input data (worksizes calculated earlier based
    // on number of GPUs
    if(worksizes == nullptr) {
        throw std::runtime_error("null worksize encountered when running FFT, was CUFFTHandle "
                                 "configured?");
    }
    throw_on_error(hipfftMakePlan1d(*handle_, num_samples, HIPFFT_C2C, 1, worksizes));

    // Copy the input data to each device
    // Note: I'm not sure of the exact reason, however HIPFFT_XT_FORMAT_INPLACE should be used
    //  instead of HIPFFT_XT_FORMAT_INPUT; doing otherwise appears to return HIPFFT_INVALID_TYPE when
    //  the following copy occurs, with the description "No longer used". The use of INPUT might be
    //  deprecated but not documented?
    hipLibXtDesc* device_data_in;
    throw_on_error(
        hipfftXtMalloc(*handle_, (hipLibXtDesc**)&device_data_in, HIPFFT_XT_FORMAT_INPLACE));
    throw_on_error(
        hipfftXtMemcpy(*handle_, device_data_in, (void*)in_data.data(), HIPFFT_COPY_HOST_TO_DEVICE));

    // Allocate space for the result on each device
    hipLibXtDesc* device_data_out;
    throw_on_error(hipfftXtMalloc(*handle_, &device_data_out, HIPFFT_XT_FORMAT_OUTPUT));

    // Execute the FFT
    throw_on_error(
        hipfftXtExecDescriptorC2C(*handle_, device_data_in, device_data_out, HIPFFT_FORWARD));

    // Create vector on host CPU using the same length as the input vector
    std::vector<float> out_data(in_data.size());

    // Copy back from device
    throw_on_error(
        hipfftXtMemcpy(*handle_, out_data.data(), device_data_out, HIPFFT_COPY_DEVICE_TO_HOST));

    // Clean up the memory we have allocated
    // TODO: Wrap these allocations in an exception safe class to avoid leaking memory if we throw
    throw_on_error(hipfftXtFree(device_data_out));
    throw_on_error(hipfftXtFree(device_data_in));

    return out_data;
}

} // namespace cuda
} // namespace cditma